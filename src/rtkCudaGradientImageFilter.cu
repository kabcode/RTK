#include "hip/hip_runtime.h"
/*=========================================================================
 *
 *  Copyright RTK Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/

/*****************
 *  rtk #includes *
 *****************/
#include "rtkCudaUtilities.hcu"
#include "rtkConfiguration.h"
#include "rtkCudaIntersectBox.hcu"
#include "rtkCudaGradientImageFilter.hcu"

/*****************
 *  C   #includes *
 *****************/
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>

/*****************
 * CUDA #includes *
 *****************/
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

__constant__ int3 c_Size;
__constant__ float3 c_Spacing;
__constant__ float c_Direction[3][3]; // 2D constant array for rotation matrices up to 3D

__global__ void gradient_kernel_2d(hipTextureObject_t in, float* grad, const int len = 2);
__global__ void gradient_kernel_3d(hipTextureObject_t in, float* grad, const int len = 3);

template <unsigned int vectorLength>
__global__ void
kernel_GradientProject(float * dev_proj_in, float * dev_proj_out, float * dev_vol, hipTextureObject_t * dev_tex_vol)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int numThread = j * c_Size.x + i;

  if (i >= c_Size.x || j >= c_Size.y)
    return;
}

template <unsigned int TDimension>
void
CUDA_gradient(
  float* dev_in,
  unsigned int* size,
  float* spacing,
  float* direction,
  unsigned int dimension,
  unsigned int boundaryCondition,
  float* dev_out)
{
  std::cout << TDimension << std::endl;

  auto addressmode = hipTextureAddressMode(boundaryCondition);
  unsigned int cSize[] = { 1,1,1 };
  float cSpacing[] = { 1,1,1 };
  float cDirection[][3] = { {1.f,0,0},{0.f,1,0},{0.f,0,1} };

  // Output volume size and spacing
  for (unsigned int i = 0; i < dimension; ++i)
  {
    cSize[i] = size[i];
    cSpacing[i] = spacing[i];
    for (unsigned int j = 0; j < dimension; ++j)
    {
      cDirection[i][j] = direction[i + j * dimension];
    }
  }

  long int outputMemorySize = cSize[0] * cSize[1] * cSize[2] * dimension * sizeof(float);
  hipMemset(dev_out, 0, outputMemorySize);

  hipMemcpyToSymbol(HIP_SYMBOL(c_Size), cSize, sizeof(int3));
  hipMemcpyToSymbol(HIP_SYMBOL(c_Spacing), cSpacing, sizeof(float3));
  hipMemcpyToSymbol(HIP_SYMBOL(c_Direction), cDirection, 3 * 3 * sizeof(float));

  switch (dimension)
  {
  case 1:
  {
    break;
  }
  case 2:
  {
    // Allocate CUDA array in device memory
    auto channelDesc = hipCreateChannelDesc<float>();
    hipArray* imgArray = nullptr;
    size_t pitch;
    hipMallocPitch(&imgArray, &pitch, cSize[1]*sizeof(float), cSize[0]);
    hipMallocArray(&imgArray, &channelDesc, cSize[0], cSize[1]);
    hipMemcpy2D(imgArray, pitch, dev_in, cSize[1]*sizeof(float), cSize[1]*sizeof(float), cSize[0], hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR;
 
    // Specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = imgArray;

    // Specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = addressmode;
    texDesc.addressMode[1] = addressmode;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    // Create texture object
    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);

    // Thread Block Dimensions
    auto dimBlock = dim3(16, 16);
    auto blocksInX = iDivUp(cSize[0], dimBlock.x);
    auto blocksInY = iDivUp(cSize[1], dimBlock.y);
    auto dimGrid = dim3(blocksInX, blocksInY);

    gradient_kernel_2d <<< dimGrid, dimBlock >>> (texObj, dev_out);
    CUDA_CHECK_ERROR;

    // Clean up
    hipDestroyTextureObject(texObj);
    hipFreeArray(imgArray);
    break;
  }
  case 3:
  {
    // Allocate CUDA array in device memory
    auto channelDesc = hipCreateChannelDesc<float>();
    auto volExtent = make_hipExtent(cSize[0], cSize[1], cSize[2]);
    hipArray* volArray = nullptr;
    hipMalloc3DArray((hipArray**)& volArray, &channelDesc, volExtent);
    hipMemcpy3DParms CopyParams = { 0 };
    CopyParams.srcPtr = make_hipPitchedPtr((void*)dev_in, cSize[0] * sizeof(float), cSize[0], cSize[1]);
    CopyParams.dstArray = volArray;
    CopyParams.extent = volExtent;
    CopyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&CopyParams);
    CUDA_CHECK_ERROR;

    // Specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = volArray;

    // Specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = addressmode;
    texDesc.addressMode[1] = addressmode;
    texDesc.addressMode[2] = addressmode;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    // Create texture object
    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);

    // Thread Block Dimensions
    dim3 dimBlock = dim3(8, 8, 8);

    int blocksInX = iDivUp(cSize[0], dimBlock.x);
    int blocksInY = iDivUp(cSize[1], dimBlock.y);
    int blocksInZ = iDivUp(cSize[2], dimBlock.z);

    dim3 dimGrid = dim3(blocksInX, blocksInY, blocksInZ);
    gradient_kernel_3d <<< dimGrid, dimBlock >>> (texObj, dev_out);
    CUDA_CHECK_ERROR;

    // Clean up
    hipDestroyTextureObject(texObj);
    hipFreeArray(volArray);
    break;
  }
  default:
    itkGenericExceptionMacro("This dimensionality is not supported.")
  }
}

__global__
void
gradient_kernel_2d(hipTextureObject_t in, float * grad, const int len)
{
  const auto i = blockIdx.x * blockDim.x + threadIdx.x;
  const auto j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i >= c_Size.x || j >= c_Size.y)
    return;

  const float _01 = tex2D<float>(in, i - 0.5, j + 0.5);
  const float _21 = tex2D<float>(in, i + 1.5, j + 0.5);
  const float _10 = tex2D<float>(in, i + 0.5, j - 0.5);
  const float _12 = tex2D<float>(in, i + 0.5, j + 1.5);

  const long int id = len * (i + c_Size.x * j);
  float grads[] = { 0.5f * (_21 - _01) / c_Spacing.x, 0.5f * (_12 - _10) / c_Spacing.y };

  for (unsigned int m = 0; m < 2; ++m)
  {
    float sum = 0.f;
    sum += c_Direction[m][0] * grads[0];
    sum += c_Direction[m][1] * grads[1];
    grad[id + m] = sum;
  }

}

__global__
void
gradient_kernel_3d(hipTextureObject_t in, float* grad, int len)
{
  const auto i = blockIdx.x * blockDim.x + threadIdx.x;
  const auto j = blockIdx.y * blockDim.y + threadIdx.y;
  const auto k = blockIdx.z * blockDim.z + threadIdx.z;

  if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z)
    return;

  //float _000 = tex3D<float>(in, i - 0.5, j - 0.5, k - 0.5);
  //float _001 = tex3D<float>(in, i - 0.5, j - 0.5, k + 0.5);
  //float _002 = tex3D<float>(in, i - 0.5, j - 0.5, k + 1.5);
  //float _010 = tex3D<float>(in, i - 0.5, j + 0.5, k - 0.5);
  const float _011 = tex3D<float>(in, i - 0.5, j + 0.5, k + 0.5);
  //float _012 = tex3D<float>(in, i - 0.5, j + 0.5, k + 1.5);
  //float _020 = tex3D<float>(in, i - 0.5, j + 1.5, k - 0.5);
  //float _021 = tex3D<float>(in, i - 0.5, j + 1.5, k + 0.5);
  //float _022 = tex3D<float>(in, i - 0.5, j + 1.5, k + 1.5);
  //float _100 = tex3D<float>(in, i + 0.5, j - 0.5, k - 0.5);
  const float _101 = tex3D<float>(in, i + 0.5, j - 0.5, k + 0.5);
  //float _102 = tex3D<float>(in, i + 0.5, j - 0.5, k + 1.5);
  const float _110 = tex3D<float>(in, i + 0.5, j + 0.5, k - 0.5);
  //float _111 = tex3D<float>(in, i + 0.5, j + 0.5, k + 0.5);
  const float _112 = tex3D<float>(in, i + 0.5, j + 0.5, k + 1.5);
  //float _120 = tex3D<float>(in, i + 0.5, j + 1.5, k - 0.5);
  const float _121 = tex3D<float>(in, i + 0.5, j + 1.5, k + 0.5);
  //float _122 = tex3D<float>(in, i + 0.5, j + 1.5, k + 1.5);
  //float _200 = tex3D<float>(in, i + 1.5, j - 0.5, k - 0.5);
  //float _201 = tex3D<float>(in, i + 1.5, j - 0.5, k + 0.5);
  //float _202 = tex3D<float>(in, i + 1.5, j - 0.5, k + 1.5);
  //float _210 = tex3D<float>(in, i + 1.5, j + 0.5, k - 0.5);
  const float _211 = tex3D<float>(in, i + 1.5, j + 0.5, k + 0.5);
  //float _212 = tex3D<float>(in, i + 1.5, j + 0.5, k + 1.5);
  //float _220 = tex3D<float>(in, i + 1.5, j + 1.5, k - 0.5);
  //float _221 = tex3D<float>(in, i + 1.5, j + 1.5, k + 0.5);
  //float _222 = tex3D<float>(in, i + 1.5, j + 1.5, k + 1.5);	
  const long int id = len * (i + c_Size.x * (j + k * c_Size.y));

  grad[id + 0] = 0.5f * (_211 - _011) / c_Spacing.x;
  grad[id + 1] = 0.5f * (_121 - _101) / c_Spacing.y;
  grad[id + 2] = 0.5f * (_112 - _110) / c_Spacing.z;
}


template void RTK_EXPORT CUDA_gradient<2>(float* dev_in, unsigned int* size, float* spacing, float* direction, unsigned int dimension, unsigned int boundaryCondition, float* dev_out);
template void RTK_EXPORT CUDA_gradient<3>(float* dev_in, unsigned int* size, float* spacing, float* direction, unsigned int dimension, unsigned int boundaryCondition, float* dev_out);

#include "hip/hip_runtime.h"
/*=========================================================================
 *
 *  Copyright RTK Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

 /*****************
  *  rtk #includes *
  *****************/
#include "rtkCudaUtilities.hcu"
#include "rtkConfiguration.h"
#include "rtkCudaResampleImageFilter.hcu"

  /*****************
   *  C   #includes *
   *****************/
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>

   /*****************
    * CUDA #includes *
    *****************/
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

template<unsigned int TDimension>
inline __device__ void multiplyMatVec(float* Mat, float* Vec, float* Return)
{
  float tmp[TDimension];
  #pragma unroll
  for (unsigned int i = 0; i < TDimension; ++i)
  {
    #pragma unroll
    for (unsigned int j = 0; j < TDimension; ++j)
    {
      tmp[i] =+ Mat[j * TDimension + i] * Vec[j];
    }
    Return[i] = tmp[i];
  }
}

template<unsigned int TDimension>
inline __device__ void multiplyScalarVec(float scalar, float* Vec, float* Return)
{
#pragma unroll
  for (unsigned int i = 0; i < TDimension; ++i)
  {
    Return[i] = scalar * Vec[i];
  }
}

template<unsigned int TDimension>
inline __device__ void cwiseVecVec(float* Vec1, float* Vec2, float* Return)
{
#pragma unroll
  for (unsigned int i = 0; i < TDimension; ++i)
  {
    Return[i] = Vec1[i] * Vec2[i];
  }
}

template<unsigned int TDimension>
inline __device__ void addVecVec(float* Vec1, float* Vec2, float* Return)
{
#pragma unroll
  for (unsigned int i = 0; i < TDimension; ++i)
  {
    Return[i] = Vec1[i] + Vec2[i];
  }
}

template<unsigned int TDimension>
inline __device__ void subVecVec(float* Vec1, float* Vec2, float* Return)
{
#pragma unroll
  for (unsigned int i = 0; i < TDimension; ++i)
  {
    Return[i] = Vec1[i] - Vec2[i];
  }
}


template<unsigned int TDimension>
__global__ void Copykernel(CudaImageProps<TDimension>* in,
  CudaImageProps<TDimension>* out,
  CudaTransformProps<TDimension, TDimension>* trans)
{
  const auto i = blockIdx.x * blockDim.x + threadIdx.x;
  const auto j = blockIdx.y * blockDim.y + threadIdx.y;
  const auto k = blockIdx.z * blockDim.z + threadIdx.z;




  if (i >= out->size[0] || j >= out->size[1] || k >= out->size[2])
    return;
  if (i == 100 && j == 50)
  {
    printf("in->direction\n[%f,%f,%f]\n[%f,%f,%f]\n[%f,%f,%f]\n",
      out->direction[0], out->direction[1], out->direction[2],
      out->direction[3], out->direction[4], out->direction[5],
      out->direction[6], out->direction[7], out->direction[8]);
  }
  float idx_out[] = { i*1.0f, j*1.0f, k*1.0f };
  // compute physical coordinates for output pixel
  float physicalOut[TDimension];
  cwiseVecVec<TDimension>(out->spacing, idx_out, physicalOut);
  if (i == 100 && j == 50) printf("physicalOut[%f,%f,%f]\n", physicalOut[0], physicalOut[1], physicalOut[2]);
  float physicalOut2[TDimension];
  float tmp[TDimension];
#pragma unroll
  for (unsigned int i = 0; i < TDimension; ++i)
  {
    tmp[i] = 0;
#pragma unroll
    for (unsigned int j = 0; j < TDimension; ++j)
    {
      tmp[i] =+ out->direction[j * TDimension + i] * physicalOut[j];
    }
    physicalOut2[i] = tmp[i];
  }
  //multiplyMatVec<TDimension>(out->direction, physicalOut, physicalOut2);
  if (i == 100 && j == 50) printf("physicalOut2[%f,%f,%f]\n", physicalOut2[0], physicalOut2[1], physicalOut2[2]);
  addVecVec<TDimension>(out->origin, physicalOut2, physicalOut);
  if (i == 100 && j == 50) printf("physicalOut[%f,%f,%f]\n", physicalOut[0], physicalOut[1], physicalOut[2]);

  // apply inverse transform towards input image

  // compute indices for physical coordinates
  float idx_in[] = { 0.f,0.f,0.f };
  subVecVec<TDimension>(physicalOut, in->origin, idx_in);
  if (i == 100 && j == 50) printf("idx_in[%f,%f,%f]\n", idx_in[0], idx_in[1], idx_in[2]);
  multiplyMatVec<TDimension>(in->pointToIndexMatrix, idx_in, idx_in);
  if (i == 100 && j == 50) printf("idx_in[%f,%f,%f]\n", idx_in[0], idx_in[1], idx_in[2]);
  /*
  if (i == 100 && j == 50)
  {
    printf("physicalOut[%f,%f,%f]\n", physicalOut[0], physicalOut[1], physicalOut[2]);
    printf("indexIn[%f,%f,%f]\n", idx_in[0], idx_in[1], idx_in[2]);
    printf("in->size[%i,%i,%i] \n", in->size[0], in->size[1], in->size[2]);
    printf("out->size[%i,%i,%i]\n", out->size[0], out->size[1], out->size[2]);
    printf("in->direction\n[%f,%f,%f]\n[%f,%f,%f]\n[%f,%f,%f]\n",
      in->direction[0], in->direction[1], in->direction[2],
      in->direction[3], in->direction[4], in->direction[5],
      in->direction[6], in->direction[7], in->direction[8]);
    printf("in->p2i\n[%f,%f,%f]\n[%f,%f,%f]\n[%f,%f,%f]\n",
      in->pointToIndexMatrix[0], in->pointToIndexMatrix[1], in->pointToIndexMatrix[2],
      in->pointToIndexMatrix[3], in->pointToIndexMatrix[4], in->pointToIndexMatrix[5],
      in->pointToIndexMatrix[6], in->pointToIndexMatrix[7], in->pointToIndexMatrix[8]);
    printf("trans->mat\n[%f,%f,%f]\n[%f,%f,%f]\n[%f,%f,%f]\n",
      trans->Matrix[0], trans->Matrix[1], trans->Matrix[2],
      trans->Matrix[3], trans->Matrix[4], trans->Matrix[5],
      trans->Matrix[6], trans->Matrix[7], trans->Matrix[8]);
    printf("trans->off[%f,%f,%f]\n", trans->Offset[0], trans->Offset[1], trans->Offset[2]);

  }
  */
  out->data[i + out->size[0] * (j + out->size[1] * k)] = tex3D<float>(in->texObj_in, idx_in[0], idx_in[1], idx_in[2]);
}

template <unsigned int TDimension>
void
CUDA_resample(
  CudaImageProps<TDimension>* h_in,
  CudaImageProps<TDimension>* h_out,
  CudaTransformProps<TDimension, TDimension>* h_trans
)
{
  CudaImageProps<TDimension>* dev_in;
  hipMalloc((void**)&dev_in, sizeof(CudaImageProps<TDimension>));
  cudaCheckErrors("hipMalloc dev_in");
  hipMemcpy(dev_in, h_in, sizeof(CudaImageProps<TDimension>), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy dev_in");
  /*
    float* h_in_dir;
    hipMalloc((void**)&h_in_dir, sizeof(float)*TDimension*TDimension);
    cudaCheckErrors("hipMalloc d_in_dir");
    hipMemcpy(h_in_dir, &h_in->direction, sizeof(float)*TDimension*TDimension, hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy h_in_dir");
    hipMemcpy(&(dev_in->direction), &h_in_dir, sizeof(float*), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy dev_in->dir");
    */
  hipMemcpy(&(dev_in->data), &(h_in->data), sizeof(float*), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy dev_in->data");

  auto channelDesc = hipCreateChannelDesc<float>();
  auto volExtent = make_hipExtent(h_in->size[0], h_in->size[1], h_in->size[2]);
  hipArray* volArray = nullptr;
  hipMalloc3DArray((hipArray**)& volArray, &channelDesc, volExtent);
  hipMemcpy3DParms CopyParams = { 0 };
  CopyParams.srcPtr = make_hipPitchedPtr((void*)h_in->data, h_in->size[0] * sizeof(float), h_in->size[0], h_in->size[1]);
  CopyParams.dstArray = volArray;
  CopyParams.extent = volExtent;
  CopyParams.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&CopyParams);
  CUDA_CHECK_ERROR;

  // Specify texture
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = volArray;

  // Specify texture object parameters
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeBorder;
  texDesc.addressMode[1] = hipAddressModeBorder;
  texDesc.addressMode[2] = hipAddressModeBorder;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  hipCreateTextureObject(&h_in->texObj_in, &resDesc, &texDesc, nullptr);
  hipMemcpy(&(dev_in->texObj_in), &(h_in->texObj_in), sizeof(hipTextureObject_t*), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy dev_in->texObj_in");

  CudaImageProps<TDimension>* dev_out;
  hipMalloc((void**)&dev_out, sizeof(CudaImageProps<TDimension>));
  cudaCheckErrors("hipMalloc dev_out");
  hipMemcpy(dev_out, h_out, sizeof(CudaImageProps<TDimension>), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy dev_out");
  hipMemcpy(&(dev_out->data), &(h_out->data), sizeof(float*), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy dev_in->data");

  CudaTransformProps<TDimension, TDimension>* dev_trans;
  hipMalloc((void**)&dev_trans, sizeof(CudaTransformProps<TDimension, TDimension>));
  cudaCheckErrors("hipMalloc dev_trans");
  hipMemcpy(dev_trans, h_trans, sizeof(CudaTransformProps<TDimension, TDimension>), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy dev_trans");

  dim3 dimBlock = dim3(16, 16, 1);
  dim3 dimGrid = dim3(iDivUp(h_in->size[0], dimBlock.x), iDivUp(h_in->size[1], dimBlock.x));

  Copykernel << <dimBlock, dimGrid >> > (dev_in, dev_out, dev_trans);
  hipDeviceSynchronize();
  cudaCheckErrors("Copykernel");

}



template void RTK_EXPORT CUDA_resample<2>(CudaImageProps<2>*, CudaImageProps<2>*, CudaTransformProps<2, 2>*);
template void RTK_EXPORT CUDA_resample<3>(CudaImageProps<3>*, CudaImageProps<3>*, CudaTransformProps<3, 3>*);

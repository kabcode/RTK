#include "hip/hip_runtime.h"
/*=========================================================================
 *
 *  Copyright RTK Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

/*****************
 *  rtk #includes *
 *****************/
#include "rtkCudaUtilities.hcu"
#include "rtkConfiguration.h"
#include "rtkCudaResampleImageFilter.hcu"

/*****************
 *  C   #includes *
 *****************/
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>

/*****************
 * CUDA #includes *
 *****************/
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

template<unsigned int TDimension>
__global__ void Copykernel(CudaImageProps<TDimension>* in, CudaImageProps<TDimension>* out)
{
  const auto i = blockIdx.x * blockDim.x + threadIdx.x;
  const auto j = blockIdx.y * blockDim.y + threadIdx.y;
  const auto k = blockIdx.z * blockDim.z + threadIdx.z;

  if(i == 1 && j == 1)
  {
    printf("in->size[%i,%i,%i] \n", in->size[0], in->size[1], in->size[2]);
    printf("out->size[%i,%i,%i]\n", out->size[0], out->size[1], out->size[2]);
  }


  if (i >= in->size[0] || j >= in->size[1] || k >= in->size[2])
    return;

  out->data[i + out->size[0] * (j + out->size[1] * k)] = in->data[i + in->size[0] * (j + in->size[1] * k)];
}

template <unsigned int TDimension>
void
CUDA_resample(
      CudaImageProps<TDimension>* h_in,
      CudaImageProps<TDimension>* h_out
)
{
  CudaImageProps<TDimension>* dev_in;
  hipMalloc((void**)&dev_in, sizeof(CudaImageProps<TDimension>));
  cudaCheckErrors("hipMalloc dev_in");
  hipMemcpy(dev_in, h_in, sizeof(CudaImageProps<TDimension>), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy dev_in");
  hipMemcpy(&(dev_in->data), &(h_in->data), sizeof(float*), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy dev_in->data");
 
  CudaImageProps<TDimension>* dev_out;
  hipMalloc((void**)&dev_out, sizeof(CudaImageProps<TDimension>));
  cudaCheckErrors("hipMalloc dev_out");
  hipMemcpy(dev_out, h_out, sizeof(CudaImageProps<TDimension>), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy dev_out");
  hipMemcpy(&(dev_out->data), &(h_out->data), sizeof(float*), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy dev_in->data");

  dim3 dimBlock = dim3(16, 16, 1);
  dim3 dimGrid = dim3(iDivUp(h_in->size[0], dimBlock.x), iDivUp(h_in->size[1], dimBlock.x));

  Copykernel<<<dimBlock,dimGrid>>>(dev_in, dev_out);
  hipDeviceSynchronize();
  cudaCheckErrors("Copykernel");

}



template void RTK_EXPORT CUDA_resample<2>(CudaImageProps<2>*,CudaImageProps<2>*);
template void RTK_EXPORT CUDA_resample<3>(CudaImageProps<3>*,CudaImageProps<3>*);

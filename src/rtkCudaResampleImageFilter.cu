#include "hip/hip_runtime.h"
/*=========================================================================
 *
 *  Copyright RTK Consortium
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

/*****************
 *  rtk #includes *
 *****************/
#include "rtkCudaUtilities.hcu"
#include "rtkConfiguration.h"
#include "rtkCudaResampleImageFilter.hcu"

/*****************
 *  C   #includes *
 *****************/
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>

/*****************
 * CUDA #includes *
 *****************/
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

template<unsigned int TDimension>
__global__ void Copykernel(CudaImageProps<TDimension>* in,
                           CudaImageProps<TDimension>* out,
                           CudaTransformProps<TDimension, TDimension>* trans)
{
  const auto i = blockIdx.x * blockDim.x + threadIdx.x;
  const auto j = blockIdx.y * blockDim.y + threadIdx.y;
  const auto k = blockIdx.z * blockDim.z + threadIdx.z;

  if(i == 1 && j == 1)
  {
    printf("in->size[%i,%i,%i] \n", in->size[0], in->size[1], in->size[2]);
    printf("out->size[%i,%i,%i]\n", out->size[0], out->size[1], out->size[2]);
    printf("trans->mat[%f,%f,%f]\n", trans->Matrix[0], trans->Matrix[1], trans->Matrix[2]);
    printf("trans->off[%f,%f,%f]\n", trans->Offset[0], trans->Offset[1], trans->Offset[2]);
  }


  if (i >= in->size[0] || j >= in->size[1] || k >= in->size[2])
    return;

  out->data[i + out->size[0] * (j + out->size[1] * k)] = tex3D<float>(in->texObj_in,i,j,k);
}

template <unsigned int TDimension>
void
CUDA_resample(
      CudaImageProps<TDimension>* h_in,
      CudaImageProps<TDimension>* h_out,
      CudaTransformProps<TDimension, TDimension>* h_trans
)
{
  CudaImageProps<TDimension>* dev_in;
  hipMalloc((void**)&dev_in, sizeof(CudaImageProps<TDimension>));
  cudaCheckErrors("hipMalloc dev_in");
  hipMemcpy(dev_in, h_in, sizeof(CudaImageProps<TDimension>), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy dev_in");
  hipMemcpy(&(dev_in->data), &(h_in->data), sizeof(float*), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy dev_in->data");

  auto channelDesc = hipCreateChannelDesc<float>();
  auto volExtent = make_hipExtent(h_in->size[0], h_in->size[1], h_in->size[2]);
  hipArray* volArray = nullptr;
  hipMalloc3DArray((hipArray**)& volArray, &channelDesc, volExtent);
  hipMemcpy3DParms CopyParams = { 0 };
  CopyParams.srcPtr = make_hipPitchedPtr((void*)h_in->data, h_in->size[0] * sizeof(float), h_in->size[0], h_in->size[1]);
  CopyParams.dstArray = volArray;
  CopyParams.extent = volExtent;
  CopyParams.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&CopyParams);
  CUDA_CHECK_ERROR;

  // Specify texture
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = volArray;

  // Specify texture object parameters
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeBorder;
  texDesc.addressMode[1] = hipAddressModeBorder;
  texDesc.addressMode[2] = hipAddressModeBorder;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  hipCreateTextureObject(&h_in->texObj_in, &resDesc, &texDesc, nullptr);
  hipMemcpy(&(dev_in->texObj_in), &(h_in->texObj_in), sizeof(hipTextureObject_t*), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy dev_in->texObj_in");
 
  CudaImageProps<TDimension>* dev_out;
  hipMalloc((void**)&dev_out, sizeof(CudaImageProps<TDimension>));
  cudaCheckErrors("hipMalloc dev_out");
  hipMemcpy(dev_out, h_out, sizeof(CudaImageProps<TDimension>), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy dev_out");
  hipMemcpy(&(dev_out->data), &(h_out->data), sizeof(float*), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy dev_in->data");

  CudaTransformProps<TDimension, TDimension>* dev_trans;
  hipMalloc((void**)&dev_trans, sizeof(CudaTransformProps<TDimension, TDimension>));
  cudaCheckErrors("hipMalloc dev_trans");
  hipMemcpy(dev_trans, h_trans, sizeof(CudaTransformProps<TDimension, TDimension>), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy dev_trans");

  dim3 dimBlock = dim3(16, 16, 1);
  dim3 dimGrid = dim3(iDivUp(h_in->size[0], dimBlock.x), iDivUp(h_in->size[1], dimBlock.x));

  Copykernel<<<dimBlock,dimGrid>>>(dev_in, dev_out, dev_trans);
  hipDeviceSynchronize();
  cudaCheckErrors("Copykernel");

}



template void RTK_EXPORT CUDA_resample<2>(CudaImageProps<2>*,CudaImageProps<2>*, CudaTransformProps<2,2>*);
template void RTK_EXPORT CUDA_resample<3>(CudaImageProps<3>*,CudaImageProps<3>*, CudaTransformProps<3,3>*);
